// including used headers

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <numeric>
#include <algorithm>
#include <random>

// kernel
__global__ void adjacent_difference(int n, float *x)
{
	// data indices to blocks
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// run algorithm
	// first element ~ do nothing
	// rest of the elements ~ compute differences
	if (i < n && i != 0)
	{
		x[i] = x[i] - x[i - 1];
	}
}

// main function
int main(int, char **)
{
	// random number generation
	std::random_device rd{};
	std::mt19937 gen(rd());
	std::normal_distribution<float> distr(-10.f, 10.f);

	auto rand = [&distr, &gen]() {
		return (float)distr(gen);
	};

	// size
	size_t size = (int)(1000 * 1000);

	// test vectors
	std::vector<float> XVec(size);
	std::generate(XVec.begin(), XVec.end(), rand);
	std::vector<float> ResultVec(XVec.size());

	// vectors for devcie
	float *devX = nullptr;

	// memory allocation on device
	hipError_t err = hipSuccess;
	err = hipMalloc((void **)&devX, size * sizeof(float));
	if (err != hipSuccess)
	{
		std::cout << "Error allocating CUDA memory (X): " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// copy data to onto device
	err = hipMemcpy(devX, XVec.data(), size * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cout << "Error copying memory to device (X): " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// grid and block dimensions
	dim3 dimGrid(size / 1000, 1);
	dim3 dimBlock(1000, 1);

	// start kernel
	adjacent_difference<<<dimGrid, dimBlock>>>((int)size, devX);

	// get errors from run
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// copy data from device
	err = hipMemcpy(ResultVec.data(), devX, size * sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		std::cout << "Error copying memory to host: " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// free memory
	err = hipFree(devX);
	if (err != hipSuccess)
	{
		std::cout << "Error freeing allocation (X): " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// write results to screen
	/*
	for (auto r : ResultVec)
	{
		std::cout << r << std::endl;
	}
	*/

	// repeat on CPU tp validate results
	std::adjacent_difference(XVec.begin(), XVec.end(), XVec.begin());

	// check equality
	if (std::equal(ResultVec.begin(), ResultVec.end(), XVec.begin()))
	{
		std::cout << "Success" << std::endl;
	}
	else
	{
		std::cout << "Mismatch between CPU and GPU results." << std::endl;
	}

	// this is the way
	return 0;
}