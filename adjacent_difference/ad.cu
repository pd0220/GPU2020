// including used headers

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <numeric>
#include <algorithm>
#include <random>

// kernel
__global__ void adjacent_difference(int n, float *x, float *y)
{
	// data indices to blocks
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// run algorithm
	// first element ~ do nothing
	// rest of the elements ~ compute differences
	if (i == 0)
	{
		y[i] = x[i];
	}
	else
	{
		y[i] = x[i] - x[i - 1];
	}
}

// main function
int main(int, char **)
{
	// random number generation
	std::random_device rd{};
	std::mt19937 gen(rd());
	std::normal_distribution<float> distr(-10.f, 10.f);
	// random generator lambda
	auto rand = [&distr, &gen]() {
		return (float)distr(gen);
	};

	// test variable
	int test = 1024;

	// size
	size_t size = (int)(test * test);

	// test vectors
	std::vector<float> XVec(size);
	std::vector<float> YVec(size);
	std::generate(XVec.begin(), XVec.end(), rand);
	std::vector<float> ResultVec(size);

	// vectors for devcie
	float *devX = nullptr;
	float *devY = nullptr;

	// memory allocation on device
	hipError_t err = hipSuccess;
	err = hipMalloc((void **)&devX, size * sizeof(float));
	if (err != hipSuccess)
	{
		std::cout << "Error allocating CUDA memory (X): " << hipGetErrorString(err) << std::endl;
		return -1;
	}
	err = hipMalloc((void **)&devY, size * sizeof(float));
	if (err != hipSuccess)
	{
		std::cout << "Error allocating CUDA memory (Y): " << hipGetErrorString(err) << std::endl;
		return -1;
	}
	
	// copy data onto device
	err = hipMemcpy(devX, XVec.data(), size * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cout << "Error copying memory to device (X): " << hipGetErrorString(err) << std::endl;
		return -1;
	}
	err = hipMemcpy(devY, YVec.data(), size * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cout << "Error copying memory to device (Y): " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// grid and block dimensions
	dim3 dimGrid(size / test, 1);
	dim3 dimBlock(test, 1);

	// start kernel
	adjacent_difference<<<dimGrid, dimBlock>>>((int)size, devX, devY);

	// get errors from run
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// copy data from device
	err = hipMemcpy(ResultVec.data(), devY, size * sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		std::cout << "Error copying memory to host: " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// free memory
	err = hipFree(devX);
	if (err != hipSuccess)
	{
		std::cout << "Error freeing allocation (X): " << hipGetErrorString(err) << std::endl;
		return -1;
	}
	err = hipFree(devY);
	if (err != hipSuccess)
	{
		std::cout << "Error freeing allocation (Y): " << hipGetErrorString(err) << std::endl;
		return -1;
	}

	// write results to screen
	/*
	for (auto r : ResultVec)
	{
		std::cout << r << std::endl;
	}
	*/

	// repeat on CPU tp validate results
	std::adjacent_difference(XVec.begin(), XVec.end(), XVec.begin());

	// check equality
	if (std::equal(ResultVec.begin(), ResultVec.end(), XVec.begin()))
	{
		std::cout << "Success" << std::endl;
	}
	else
	{
		std::cout << "Mismatch between CPU and GPU results." << std::endl;
	}

	// this is the way
	return 0;
}